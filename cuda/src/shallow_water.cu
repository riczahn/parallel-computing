#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_cooperative_groups.h>

#include "../inc/argument_utils.h"

namespace cg = cooperative_groups;

typedef int64_t int_t;
typedef double real_t;

int_t
    N,
    max_iteration,
    snapshot_frequency;

const real_t
    domain_size = 10.0,
    gravity = 9.81,
    density = 997.0;

real_t
    *h_mass_0 = NULL,
    *h_mass_1 = NULL,
    *d_mass_0 = NULL,
    *d_mass_1 = NULL,

    *h_mass_velocity_x_0 = NULL,
    *h_mass_velocity_x_1 = NULL,
    *d_mass_velocity_x_0 = NULL,
    *d_mass_velocity_x_1 = NULL,

    *h_mass_velocity_y_0 = NULL,
    *h_mass_velocity_y_1 = NULL,
    *d_mass_velocity_y_0 = NULL,
    *d_mass_velocity_y_1 = NULL,

    *h_mass_velocity = NULL,
    *d_mass_velocity = NULL,

    *h_velocity_x = NULL,
    *d_velocity_x = NULL,
    *h_velocity_y = NULL,
    *d_velocity_y = NULL,

    *h_acceleration_x = NULL,
    *d_acceleration_x = NULL,
    *h_acceleration_y = NULL,
    *d_acceleration_y = NULL,
    dx,
    dt;

#define PN(y,x)         mass_0[(y)*(N+2)+(x)]
#define PN_next(y,x)    mass_1[(y)*(N+2)+(x)]
#define PNU(y,x)        mass_velocity_x_0[(y)*(N+2)+(x)]
#define PNU_next(y,x)   mass_velocity_x_1[(y)*(N+2)+(x)]
#define PNV(y,x)        mass_velocity_y_0[(y)*(N+2)+(x)]
#define PNV_next(y,x)   mass_velocity_y_1[(y)*(N+2)+(x)]
#define PNUV(y,x)       mass_velocity[(y)*(N+2)+(x)]
#define U(y,x)          velocity_x[(y)*(N+2)+(x)]
#define V(y,x)          velocity_y[(y)*(N+2)+(x)]
#define DU(y,x)         acceleration_x[(y)*(N+2)+(x)]
#define DV(y,x)         acceleration_y[(y)*(N+2)+(x)]


#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void time_step(
    real_t *velocity_x,
    real_t *velocity_y,
    real_t *acceleration_x,
    real_t *acceleration_y,
	real_t *mass_velocity_x_0,
    real_t *mass_velocity_x_1,
    real_t *mass_velocity_y_0,
    real_t *mass_velocity_y_1,
	real_t *mass_velocity,
    real_t *mass_0,
    real_t *mass_1,
    int_t N,
    real_t dx,
    real_t dt
);

__device__ void time_step_1(
    real_t *velocity_x,
    real_t *mass_velocity_x_0,
    real_t *mass_0,
    real_t *velocity_y,
    real_t *mass_velocity_y_0,
    real_t *acceleration_x,
    real_t *acceleration_y,
    real_t *mass_velocity,
    int_t N
);

__device__ void time_step_2(
    real_t *velocity_x,
    real_t *velocity_y,
    real_t *acceleration_x,
    real_t *acceleration_y,
	real_t *mass_velocity_x_0,
    real_t *mass_velocity_x_1,
    real_t *mass_velocity_y_0,
    real_t *mass_velocity_y_1,
	real_t *mass_velocity,
    real_t *mass_0,
    real_t *mass_1,
    int_t N,
    real_t dx,
    real_t dt
);

// TODO: Rewrite boundary_condition as a device function.
__device__ void boundary_condition(real_t *domain_variable, int sign, int_t N);
void domain_init(void);
void domain_save(int_t iteration);
void domain_finalize(void);

// Pthreads threaded domain save function
void *domain_save_threaded(void *iter);

void swap(real_t** t1, real_t** t2)
{
    real_t* tmp;
	tmp = *t1;
	*t1 = *t2;
	*t2 = tmp;
}


int main (int argc, char **argv)
{

    OPTIONS *options = parse_args(argc, argv);
    if(!options)
    {
        fprintf( stderr, "Argument parsing failed\n" );
        exit(1);
    }

    N = options->N;
    max_iteration = options->max_iteration;
    snapshot_frequency = options->snapshot_frequency;

    domain_init();

    dim3 threadBlockDims;
    dim3 gridDims;

    if((N+2) <= 32) {
        threadBlockDims = {(unsigned) N+2, (unsigned) N+2, 1};
        gridDims = {1, 1, 1};
    } else {
        threadBlockDims = {32, 32, 1};
        unsigned int size = ceil(((N+2)/32.0));

        gridDims = {size, size, 1};
    }

    printf("N: %ld, SIZE: %d\n", N, gridDims.x);

    for ( int_t iteration = 0; iteration <= max_iteration; iteration++ )
    {
        /*
        // Launch the cooperative kernel
        void *kernelArgs[] = {
            d_velocity_x,
            d_velocity_y,
			d_acceleration_x,
            d_acceleration_y,
			d_mass_velocity_x_0,
            d_mass_velocity_x_1,
			d_mass_velocity_y_0,
            d_mass_velocity_y_1,
			d_mass_velocity,
            d_mass_0,
            d_mass_1,
            &N,
            &dx,
            &dt
        };

        hipError_t error = hipLaunchCooperativeKernel((void*)time_step, gridDims, threadBlockDims, kernelArgs);
        // printf("N: %d\n", error);
        */
        
        // launch the kernel
        time_step<<<gridDims, threadBlockDims>>>(
            d_velocity_x,
            d_velocity_y,
			d_acceleration_x,
            d_acceleration_y,
			d_mass_velocity_x_0,
            d_mass_velocity_x_1,
			d_mass_velocity_y_0,
            d_mass_velocity_y_1,
			d_mass_velocity,
            d_mass_0,
            d_mass_1,
            N,
            dx,
            dt
        );
        

        if(iteration % snapshot_frequency == 0)
        {
            printf(
                "Iteration %ld of %ld, (%.2lf%% complete)\n",
                iteration,
                max_iteration,
                100.0 * (real_t) iteration / (real_t) max_iteration
            );

            // copy the masses from the device to host prior to domain_save
            hipMemcpy(h_mass_0, d_mass_0, (N+2) * (N+2) * sizeof(real_t), hipMemcpyDeviceToHost);

            domain_save(iteration);
        }

        // swap device buffer pointers between iterations
        swap(&d_mass_0, &d_mass_1);
        swap(&d_mass_velocity_x_0, &d_mass_velocity_x_1);
        swap(&d_mass_velocity_y_0, &d_mass_velocity_y_1);
    }

    domain_finalize();

    exit(EXIT_SUCCESS);
}

// performs a full time step
__global__ void time_step(
    real_t *velocity_x,
    real_t *velocity_y,
    real_t *acceleration_x,
    real_t *acceleration_y,
	real_t *mass_velocity_x_0,
    real_t *mass_velocity_x_1,
    real_t *mass_velocity_y_0,
    real_t *mass_velocity_y_1,
	real_t *mass_velocity,
    real_t *mass_0,
    real_t *mass_1,
    int_t N,
    real_t dx,
    real_t dt
)
{
    // would be used if cooperative kernel would work :<
    cg::grid_group grid = cg::this_grid();

    time_step_1(
        velocity_x,
        mass_velocity_x_0,
        mass_0,
        velocity_y,
        mass_velocity_y_0,
        acceleration_x,
        acceleration_y,
        mass_velocity,
        N
    );

    // sync the device before continuing with time_step_2

    __syncthreads();
    // grid.sync();

    time_step_2(
        velocity_x,
        velocity_y,
        acceleration_x,
        acceleration_y,
        mass_velocity_x_0,
        mass_velocity_x_1,
        mass_velocity_y_0,
        mass_velocity_y_1,
        mass_velocity,
        mass_0,
        mass_1,
        N,
        dx,
        dt
    );
}

// performs the update for the accelerations
__device__ void time_step_1(
    real_t *velocity_x,
    real_t *mass_velocity_x_0,
    real_t *mass_0,
    real_t *velocity_y,
    real_t *mass_velocity_y_0,
    real_t *acceleration_x,
    real_t *acceleration_y,
    real_t *mass_velocity,
    int_t N
)
{
    boundary_condition(mass_0, 1, N);
    boundary_condition(mass_velocity_x_0, -1, N);
    boundary_condition(mass_velocity_y_0, -1, N);

    // calculate the thread id
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    // guard to not access data out of bound
    if(!(x > N || y > N || x < 1 || y < 1)) {
        U(y,x) = PNU(y,x) / PN(y,x);
        V(y,x) = PNV(y,x) / PN(y,x);

        PNUV(y,x) = PN(y,x) * U(y,x) * V(y,x);
    }

    if(!(x > N+1 || y > N+1 || x < 0 || y < 0)) {
        DU(y,x) = PN(y,x) * U(y,x) * U(y,x)
                + 0.5 * gravity * ( PN(y,x) * PN(y,x) / density );
        DV(y,x) = PN(y,x) * V(y,x) * V(y,x)
                + 0.5 * gravity * ( PN(y,x) * PN(y,x) / density );
    }
}

// performs the rest of the time step
__device__ void time_step_2(
    real_t *velocity_x,
    real_t *velocity_y,
    real_t *acceleration_x,
    real_t *acceleration_y,
	real_t *mass_velocity_x_0,
    real_t *mass_velocity_x_1,
    real_t *mass_velocity_y_0,
    real_t *mass_velocity_y_1,
	real_t *mass_velocity,
    real_t *mass_0,
    real_t *mass_1,
    int_t N,
    real_t dx,
    real_t dt
)
{
    // calculate the thread id
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if(!(x > N || y > N || x < 1 || y < 1)) {
        PNU_next(y,x) = 0.5*( PNU(y,x+1) + PNU(y,x-1) ) - dt*(
                        ( DU(y,x+1) - DU(y,x-1) ) / (2*dx)
                      + ( PNUV(y,x+1) - PNUV(y,x-1) ) / (2*dx)
        );

        PNV_next(y,x) = 0.5*( PNV(y+1,x) + PNV(y-1,x) ) - dt*(
                        ( DV(y+1,x) - DV(y-1,x) ) / (2*dx)
                      + ( PNUV(y+1,x) - PNUV(y-1,x) ) / (2*dx)
        );

        PN_next(y,x) = 0.25*( PN(y,x+1) + PN(y,x-1) + PN(y+1,x) + PN(y-1,x) ) - dt*(
                        ( PNU(y,x+1) - PNU(y,x-1) ) / (2*dx)
                      + ( PNV(y+1,x) - PNV(y-1,x) ) / (2*dx)
        );
    }
}

__device__ void boundary_condition(real_t *domain_variable, int sign, int_t N)
{
    // calculate the thread id
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    #define VAR(y,x) domain_variable[(y)*(N+2)+(x)]
    if(x == 0) {
        if(y == 0) {
            // corner (0,0)
            VAR(   0, 0   ) = sign*VAR(   2, 2   );
            return;
        }

        if(y == N+1) {
            // corner (N+1, 0)
            VAR( N+1, 0   ) = sign*VAR( N-1, 2   );
            return;
        }

        // first column from the left
        VAR(   y, 0   ) = sign*VAR(   y, 2   );
    }

    if(x == N+1) {
        if(y == 0) {
            // corner (0, N+1)
            VAR(   0, N+1 ) = sign*VAR(   2, N-1 );
            return;
        }

        if(y == N+1) {
            // corner (N+1, N+1)
            VAR( N+1, N+1 ) = sign*VAR( N-1, N-1 );
            return;
        }

        // last column on the right
        VAR(   y, N+1 ) = sign*VAR(   y, N-1 );
    }

    if(y == 0) {
        // first row
        VAR(   0, x   ) = sign*VAR(   2, x   );
    }

    if(y == N+1) {
        // last row
        VAR( N+1, x   ) = sign*VAR( N-1, x   );
    }

    #undef VAR
}


void domain_init(void)
{
    int elements = (N+2)*(N+2);

    // Allocate memory for the host buffers
    h_mass_0 = (real_t *) calloc(elements, sizeof(real_t));
    h_mass_1 = (real_t *) calloc(elements, sizeof(real_t));

    h_mass_velocity_x_0 = (real_t *) calloc(elements, sizeof(real_t));
    h_mass_velocity_x_1 = (real_t *) calloc(elements, sizeof(real_t));
    h_mass_velocity_y_0 = (real_t *) calloc(elements, sizeof(real_t));
    h_mass_velocity_y_1 = (real_t *) calloc(elements, sizeof(real_t));

    h_mass_velocity = (real_t *) calloc(elements, sizeof(real_t));

    h_velocity_x = (real_t *) calloc(elements, sizeof(real_t));
    h_velocity_y = (real_t *) calloc(elements, sizeof(real_t));
    h_acceleration_x = (real_t *) calloc(elements, sizeof(real_t));
    h_acceleration_y = (real_t *) calloc(elements, sizeof(real_t));

    // Allocate memory for the device buffers
    hipMalloc(&d_mass_0, elements * sizeof(real_t));
    hipMalloc(&d_mass_1, elements * sizeof(real_t));

    hipMalloc(&d_mass_velocity_x_0, elements * sizeof(real_t));
    hipMalloc(&d_mass_velocity_x_1, elements * sizeof(real_t));
    hipMalloc(&d_mass_velocity_y_0, elements * sizeof(real_t));
    hipMalloc(&d_mass_velocity_y_1, elements * sizeof(real_t));

    hipMalloc(&d_mass_velocity, elements * sizeof(real_t));

    hipMalloc(&d_velocity_x, elements * sizeof(real_t));
    hipMalloc(&d_velocity_y, elements * sizeof(real_t));
    hipMalloc(&d_acceleration_x, elements * sizeof(real_t));
    hipMalloc(&d_acceleration_y, elements * sizeof(real_t));

    for(int_t y=1; y<=N; y++)
    {
        for(int_t x=1; x<=N; x++)
        {
	    h_mass_0[y*(N+2) + x] = 1e-3;
	    h_mass_velocity_x_0[y*(N+2) + x] = 0.0;
	    h_mass_velocity_y_0[y*(N+2) + x] = 0.0;

            real_t cx = x-N/2;
            real_t cy = y-N/2;
            if(sqrt(cx*cx + cy*cy) < N/20.0)
            {
                h_mass_0[y*(N+2) + x] -= 5e-4*exp (
                    - 4*pow(cx, 2.0) / (real_t)(N)
                    - 4*pow(cy, 2.0) / (real_t)(N)
                );
            }

            h_mass_0[y*(N+2) + x] *= density;
        }
    }

    dx = domain_size / (real_t) N;
    dt = 5e-2;

    // copy data over to device buffer
    hipMemcpy(d_mass_0, h_mass_0, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_velocity_x_0, h_mass_velocity_x_0, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_velocity_y_0, h_mass_velocity_y_0, elements * sizeof(real_t), hipMemcpyHostToDevice);

    // also copy the zeros over
    hipMemcpy(d_mass_1, h_mass_1, elements * sizeof(real_t), hipMemcpyHostToDevice);

    hipMemcpy(d_mass_velocity_x_1, h_mass_velocity_x_1, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_velocity_y_1, h_mass_velocity_y_1, elements * sizeof(real_t), hipMemcpyHostToDevice);

    hipMemcpy(d_mass_velocity, h_mass_velocity, elements * sizeof(real_t), hipMemcpyHostToDevice);

    hipMemcpy(d_velocity_x, h_velocity_x, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_velocity_y, h_velocity_y, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_acceleration_x, h_acceleration_x, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_acceleration_y, h_acceleration_y, elements * sizeof(real_t), hipMemcpyHostToDevice);
}


void domain_save(int_t iteration)
{
    int_t index = iteration / snapshot_frequency;
    char filename[256];
    memset(filename, 0, 256*sizeof(char));
    sprintf(filename, "data/%.5ld.bin", index);

    FILE *out = fopen(filename, "wb");
    if(!out)
    {
        fprintf(stderr, "Failed to open file %s\n", filename);
        exit(1);
    }
    //fwrite ( mass[0], (N+2)*(N+2), sizeof(real_t), out );
    for(int_t y = 1; y <= N; y++)
    {
        fwrite(&h_mass_0[y*(N+2)+1], N, sizeof(real_t), out);
    }
    fclose(out);
}

void domain_finalize(void)
{
    free(h_mass_0);
    free(h_mass_1);
    free(h_mass_velocity_x_0);
    free(h_mass_velocity_x_1);
    free(h_mass_velocity_y_0);
    free(h_mass_velocity_y_1);
    free(h_mass_velocity);
    free(h_velocity_x);
    free(h_velocity_y);
    free(h_acceleration_x);
    free(h_acceleration_y);

    // Free device arrays
    hipFree(d_mass_0);
    hipFree(d_mass_1);
    hipFree(d_mass_velocity_x_0);
    hipFree(d_mass_velocity_x_1);
    hipFree(d_mass_velocity_y_0);
    hipFree(d_mass_velocity_y_1);
    hipFree(d_mass_velocity);
    hipFree(d_velocity_x);
    hipFree(d_velocity_y);
    hipFree(d_acceleration_x);
    hipFree(d_acceleration_y);
}
